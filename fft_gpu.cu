#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <stdio.h>

#define FFT_SIZE 8

void print_fft(hipfftComplex *X, int N, const char *title) {
    printf("\n%s:\n", title);
    for (int i = 0; i < N; i++) {
        printf("X[%d] = %.5f + %.5fi\n", i, X[i].x, X[i].y);
    }
}

int main() {
    hipfftComplex h_X[FFT_SIZE];

    for (int i = 0; i < FFT_SIZE; i++) {
        h_X[i].x = (i < FFT_SIZE / 2) ? 1.0 : 0.0;
        h_X[i].y = 0.0;
    }

    print_fft(h_X, FFT_SIZE, "Sinal Original");

    hipfftComplex *d_X;
    hipMalloc((void**)&d_X, sizeof(hipfftComplex) * FFT_SIZE);
    hipMemcpy(d_X, h_X, sizeof(hipfftComplex) * FFT_SIZE, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, FFT_SIZE, HIPFFT_C2C, 1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    hipfftExecC2C(plan, d_X, d_X, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    hipMemcpy(h_X, d_X, sizeof(hipfftComplex) * FFT_SIZE, hipMemcpyDeviceToHost);
    hipfftDestroy(plan);
    hipFree(d_X);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    print_fft(h_X, FFT_SIZE, "Resultado FFT na GPU");
    printf("Tempo de execução (GPU): %.6f milissegundos\n", milliseconds);

    return 0;
}
